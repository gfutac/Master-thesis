#include "hip/hip_runtime.h"
#include <opencv2\gpu\devmem2d.hpp>

#include <hip/hip_runtime.h>

#include <thrust\device_vector.h>

using namespace cv::gpu;
using namespace thrust;

#define N 4096
#define VECTORS 169
#define PROJECTIONS 19 // templates
#define SIZE 151

__global__ void calculateDistances(DevMem2Df eigens, DevMem2Df projections, DevMem2Df means, DevMem2Di samples, float *distances){
	int idx = blockIdx.x;
	int tid = threadIdx.x;
	
	int i, j, k;
	i = j = k = 0;

	__shared__ float p[VECTORS * PROJECTIONS];

	if (tid == 0){
		for (i = 0; i < VECTORS * PROJECTIONS; ++i) {
			p[i] = projections(idx)[i];
		}
	}

	__syncthreads();

	float *eig = eigens(idx);
	float *mean = means(idx);

	float sample[N];


	for (i = 0; i < N; i += 8){
		sample[i] = samples(tid)[i] - mean[i];
		sample[i+1] = samples(tid)[i+1] - mean[i+1];
		sample[i+2] = samples(tid)[i+2] - mean[i+2];
		sample[i+3] = samples(tid)[i+3] - mean[i+3];
		sample[i+4] = samples(tid)[i+4] - mean[i+4];
		sample[i+5] = samples(tid)[i+5] - mean[i+5];
		sample[i+6] = samples(tid)[i+6] - mean[i+6];
		sample[i+7] = samples(tid)[i+7] - mean[i+7];
	}

	// projicirati uzorak
	float proj[VECTORS];
	for (i = 0; i < VECTORS; ++i) {
		proj[i] = 0;
		for (j = 0; j < N; j += 8) {
			proj[i] += sample[j] * eig[i * N + j] +
				sample[j+1] * eig[i * N + j+1] + 
				sample[j+2] * eig[i * N + j+2] +
				sample[j+3] * eig[i * N + j+3] +
				sample[j+4] * eig[i * N + j+4] +
				sample[j+5] * eig[i * N + j+5] + 
				sample[j+6] * eig[i * N + j+6] +
				sample[j+7] * eig[i * N + j+7];
				//sample[j+8] * eig[i * N + j+8] +
				//sample[j+9] * eig[i * N + j+9] +
				//sample[j+10] * eig[i * N + j+10] +
				//sample[j+11] * eig[i * N + j+11] +
				//sample[j+12] * eig[i * N + j+12] +
				//sample[j+13] * eig[i * N + j+13] +
				//sample[j+14] * eig[i * N + j+14] +
				//sample[j+15] * eig[i * N + j+15];
		}
	}

	// usporediti sa svakim projiciranim i dobiti najmanju udaljenost
	float min = FLT_MAX;
	for (i = 0; i < PROJECTIONS; ++i) {
		float res = 0;
		for (j = 0; j < VECTORS; ++j) {
			//res = res + fabsf(proj[j] - p[i * VECTORS + j]);
			res += (proj[j] - p[i * VECTORS + j]) * (proj[j] - p[i * VECTORS + j]);
		}

		res = sqrtf(res);
		if (res < min) min = res;
	}

	distances[tid * SIZE + idx] = min;

}

extern "C" void getMinDistances(DevMem2Df eigens, DevMem2Df projections, DevMem2Df means, DevMem2Di samples, float *distances, int size, int *minIndices){
	float *dsample, *d_distances; 
	int *d_minindices;

	hipMalloc((float **)&d_distances, sizeof(float) * size * size);
	hipMalloc((float **)&dsample, sizeof(float) * N);
	hipMalloc((int **)&d_minindices, sizeof(int) * size);

	hipEvent_t event1, event2;
	hipEventCreate(&event1);
	hipEventCreate(&event2);


		//record events around kernel launch
		hipEventRecord(event1, 0); //where 0 is the default stream
		calculateDistances<<<size, size>>>(eigens, projections, means, samples, d_distances);
		hipEventRecord(event2, 0);

		//synchronize
		hipEventSynchronize(event1); //optional
		hipEventSynchronize(event2); //wait for the event to be executed!

		//calculate time
		float dt_ms;
		hipEventElapsedTime(&dt_ms, event1, event2);

		std::cout << i << " " << (dt_ms/1000) << std::endl;	

	//hipMemcpy(minIndices, d_minindices, sizeof(int) * size, hipMemcpyDeviceToHost);

	hipMemcpy(distances, d_distances, sizeof(float) * size * size, hipMemcpyDeviceToHost);
	
	hipFree(d_distances);
	hipFree(dsample);
}
